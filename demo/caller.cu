#include "hip/hip_runtime.h"
#include "your_api.h"

constexpr int DIM = 64;

int main() {
  hipStream_t stream;
  hipStreamCreate(&stream);
  HashTable<int64_t, float, DIM> table;

  table.insert(0, nullptr, nullptr, stream);
  table.find(0, nullptr, nullptr, nullptr, stream);

  hipStreamDestroy(stream);
  return 0;
}
